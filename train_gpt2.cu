#include "hip/hip_runtime.h"
/*
GPT-2 Transformer Neural Net training loop. See README.md for usage.
*/

#include <unistd.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdarg.h>
#include <string>
#include <string_view>
#include <sys/stat.h>
#include <sys/types.h>
// ----------- CPU utilities -----------
// defines: fopenCheck, freadCheck, fcloseCheck, fseekCheck, mallocCheck
// defines: create_dir_if_not_exists, find_max_step
#include "llmc/utils.h"
// defines: tokenizer_init, tokenizer_decode, tokenizer_free
#include "llmc/tokenizer.h"
// defines: dataloader_init, dataloader_reset, dataloader_next_batch, dataloader_free
// defines: evalloader_init, evalloader_reset, evalloader_next_batch, evalloader_free
#include "llmc/dataloader.h"
// defines: manual_seed, normal_ (same as torch.manual_seed and torch.normal)
#include "llmc/rand.h"
// defines: sample_softmax, random_f32
#include "llmc/sampler.h"
// defines: logger_init, logger_log_eval, logger_log_val, logger_log_train
#include "llmc/logger.h"
// defines: get_flops_promised
#include "llmc/mfu.h"
// ----------- GPU utilities -----------
// defines:
// WARP_SIZE, MAX_1024_THREADS_BLOCKS, CEIL_DIV, cudaCheck, PRECISION_MODE
// NVTX_RANGE_FN
#include "llmc/cuda_common.h"
// defines:
// Packed128, f128, x128
// warpReduceSum, warpReduceMax, blockReduce, copy_and_cast_kernel
#include "llmc/cuda_utils.cuh"
// defines: CUBLAS_LOWP, cublasCheck, cublaslt_workspace_size, cublaslt_workspace
// defines: cublas_compute, cublaslt_handle, cublas_handle
#include "llmc/cublas_common.h"
// ----------- Layer implementations in CUDA -----------
// defines: encoder_forward, encoder_backward
#include "llmc/encoder.cuh"
// defines: layernorm_forward, residual_forward, fused_residual_forward5, layernorm_backward
#include "llmc/layernorm.cuh"
// defines: gelu_forward, gelu_backward_inplace
#include "llmc/gelu.cuh"
#ifdef ENABLE_CUDNN
// defines: create_cudnn, destroy_cudnn, attention_forward_cudnn, attention_backward_cudnn
#include "llmc/cudnn_att.h"
#else
// defines: attention_forward, attention_backward
#include "llmc/attention.cuh"
#endif
// defines: matmul_forward, matmul_backward
#include "llmc/matmul.cuh"
// defines: fused_classifier
#include "llmc/fused_classifier.cuh"
// defines: adamw_kernel3
#include "llmc/adamw.cuh"
// defines: global_norm_squared
#include "llmc/global_norm.cuh"
// ----------- Multi-GPU support -----------
#ifdef MULTI_GPU
#include <mpi.h>
#include <nccl.h>
#endif

// ----------------------------------------------------------------------------
// global vars containing information about the GPU this process is running on
hipDeviceProp_t deviceProp; // fills in common_start()
hipStream_t main_stream;

// ----------------------------------------------------------------------------
// Multi-GPU related
#ifdef MULTI_GPU

#if defined(ENABLE_FP32)
const ncclDataType_t ncclFloatX = ncclFloat;
#elif defined(ENABLE_FP16)
const ncclDataType_t ncclFloatX = ncclHalf;
#else // Default to bfloat16
const ncclDataType_t ncclFloatX = ncclBfloat16;
#endif

void nccl_check(ncclResult_t status, const char *file, int line) {
    if (status != ncclSuccess) {
        printf("[NCCL ERROR] at file %s:%d:\n%s\n", file, line, ncclGetErrorString(status));
        exit(EXIT_FAILURE);
    }
}
#define ncclCheck(err) (nccl_check(err, __FILE__, __LINE__))

void mpi_check(int status, const char *file, int line) {
    if (status != MPI_SUCCESS) {
        char mpi_error[4096];
        int mpi_error_len = 0;
        assert(MPI_Error_string(status, &mpi_error[0], &mpi_error_len) == MPI_SUCCESS);
        printf("[MPI ERROR] at file %s:%d:\n%.*s\n", file, line, mpi_error_len, mpi_error);
        exit(EXIT_FAILURE);
    }
}
#define mpiCheck(err) (mpi_check(err, __FILE__, __LINE__))

#endif // MULTI_GPU

// ----------------------------------------------------------------------------
// MPI / multi-processing setup

// Parameters specific to training on multiple GPUs.
typedef struct {
    int process_rank;      // Rank of this process among all MPI processes. 0 if no multi-GPU.
    int num_processes;     // Total number of processes. 1 if no multi-GPU.
    int local_device_idx;  // This process GPU index on current machine. 0 if no multi-GPU.

    // Zero Redundancy Optimizer stage - https://fairscale.readthedocs.io/en/stable/deep_dive/oss_sdp_fsdp.html
    // 0-Disabled
    // 1-Optimizer State Sharding (OSS)
    // 2-Optimizer + Gradient State Sharding (SDP)
    // 3-Optimizer + Gradient + Horizontal Model Sharding (FSDP)
    int zero_stage;
    size_t shard_num_parameters;
    size_t shard_offset;
#ifdef MULTI_GPU
    ncclComm_t nccl_comm;  // NCCL communication primitive, used for collective multi-GPU work.
#endif
} MultiGpuConfig;

// one global variable to hold the multi-GPU configuration for this process
MultiGpuConfig multi_gpu_config;

#ifdef MULTI_GPU
// Determine which GPU this process should use.
// Processes on the same machines use different GPU indicies. Processes on other machines don't.
// Copied from NCCL examples: https://docs.nvidia.com/deeplearning/nccl/user-guide/docs/examples.html#example-2-one-device-per-process-or-thread
int multi_gpu_get_local_device_idx(int process_rank, int num_processes) {
  char hostname[1024];
  hostname[1023] = '\0';
  // All processes on the same machine will share the same hostname.
  gethostname(hostname, 1023);
  for (int i=0; i < 1024; i++) {
    if (hostname[i] == '.') {
        hostname[i] = '\0';
        break;
    }
  }
  uint64_t hostname_hash = 5381;
  for (int c = 0; hostname[c] != '\0'; c++){ hostname_hash = ((hostname_hash << 5) + hostname_hash) ^ hostname[c]; }

  // Distribute all hostname hashes to all processes.
  uint64_t* all_hostsname_hashes = (uint64_t*)malloc(num_processes * sizeof(uint64_t));
  all_hostsname_hashes[process_rank] = hostname_hash;
  mpiCheck(MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, all_hostsname_hashes, sizeof(uint64_t), MPI_BYTE, MPI_COMM_WORLD));

  // Identify which GPU we need to use.
  int local_device_idx = 0;
  for (int current_process = 0; current_process < num_processes; ++current_process) {
     if (current_process == process_rank) {
      // Found my gpu, local_device_idx now has my target GPU index.
      break;
     }
     if (all_hostsname_hashes[current_process] == all_hostsname_hashes[process_rank]) {
      // This process ID runs on the same machine, but it's not me, skip this GPU
      local_device_idx++;
     }
  }

  free(all_hostsname_hashes);
  return local_device_idx;
}
#endif

MultiGpuConfig multi_gpu_config_init(int *argc, char ***argv) {
#ifdef MULTI_GPU
    // Initialize MPI.
    MultiGpuConfig result;
    mpiCheck(MPI_Init(argc, argv));
    mpiCheck(MPI_Comm_rank(MPI_COMM_WORLD, &result.process_rank));
    mpiCheck(MPI_Comm_size(MPI_COMM_WORLD, &result.num_processes));
    result.local_device_idx = multi_gpu_get_local_device_idx(result.process_rank, result.num_processes);
    cudaCheck(hipSetDevice(result.local_device_idx));
    ncclUniqueId nccl_id;
    if (result.process_rank == 0) {
        ncclCheck(ncclGetUniqueId(&nccl_id));
    }
    mpiCheck(MPI_Bcast((void *)&nccl_id, sizeof(nccl_id), MPI_BYTE, 0, MPI_COMM_WORLD));
    ncclCheck(ncclCommInitRank(&result.nccl_comm, result.num_processes, nccl_id, result.process_rank));
    return result;
#else
    printf("Multi-GPU support is disabled. Using a single GPU.\n");
    cudaCheck(hipSetDevice(0));
    MultiGpuConfig result;
    result.process_rank = 0;
    result.num_processes = 1;
    result.local_device_idx = 0;
    return result;
#endif
}

void multi_gpu_config_free(const MultiGpuConfig* multi_gpu_config) {
#ifdef MULTI_GPU
    ncclCheck(ncclCommDestroy(multi_gpu_config->nccl_comm));
    mpiCheck(MPI_Finalize());
#endif
}

void multi_gpu_barrier(const MultiGpuConfig* multi_gpu_config) {
#ifdef MULTI_GPU
    if (multi_gpu_config->num_processes > 1) {
        mpiCheck(MPI_Barrier(MPI_COMM_WORLD));
    }
#endif
}

// convenience function that only prints if the rank of process is zero
void printf0(const char *format, ...) {
    if (multi_gpu_config.process_rank == 0) {
        va_list args;
        va_start(args, format);
        vprintf(format, args);
        va_end(args);
    }
}

void set_zero_configs(MultiGpuConfig* multi_gpu_config, int zero_stage, size_t total_parameters) {

    multi_gpu_config->zero_stage = 0;
    multi_gpu_config->shard_num_parameters = total_parameters;
    multi_gpu_config->shard_offset = 0;

    // Check the Zero Stage and define sharding parameters
    if (zero_stage == 0) {
        printf0("| Zero Optimization is disabled                                              |\n");
    }
    else if (zero_stage == 1) {
        if (total_parameters % multi_gpu_config->num_processes != 0) {
            printf0("| Zero Optimization is disabled, Can't equally partition parameters          |\n");
            multi_gpu_config->zero_stage = 0;
        }
        else {
            printf0("| Zero Stage1 is enabled                                                     |\n");
            multi_gpu_config->zero_stage = 1;
            multi_gpu_config->shard_num_parameters = total_parameters / multi_gpu_config->num_processes;
            multi_gpu_config->shard_offset = multi_gpu_config->process_rank * multi_gpu_config->shard_num_parameters;
        }
    }
    else{
        printf0("| Disabling Zero Optimization, Zero Stage2 and Stage3 are not yet supported  |\n");
        multi_gpu_config->zero_stage = 0;
    }
}

// ----------------------------------------------------------------------------
// GPT-2 model definition

typedef struct {
    int max_seq_len; // max sequence length, e.g. 1024
    int vocab_size; // vocab size, e.g. 50257
    int padded_vocab_size; // padded to e.g. %128==0, 50304
    int num_layers; // number of layers, e.g. 12
    int num_heads; // number of heads in attention, e.g. 12
    int channels; // number of channels, e.g. 768
} GPT2Config;

// the parameters of the model
constexpr const int NUM_PARAMETER_TENSORS = 16;
typedef struct {
    floatX* wte; // (V, C)
    floatX* wpe; // (maxT, C)
    floatX* ln1w; // (L, C)
    floatX* ln1b; // (L, C)
    floatX* qkvw; // (L, 3*C, C)
    floatX* qkvb; // (L, 3*C)
    floatX* attprojw; // (L, C, C)
    floatX* attprojb; // (L, C)
    floatX* ln2w; // (L, C)
    floatX* ln2b; // (L, C)
    floatX* fcw; // (L, 4*C, C)
    floatX* fcb; // (L, 4*C)
    floatX* fcprojw; // (L, C, 4*C)
    floatX* fcprojb; // (L, C)
    floatX* lnfw; // (C)
    floatX* lnfb; // (C)
} ParameterTensors;
static_assert(sizeof(ParameterTensors) == NUM_PARAMETER_TENSORS * sizeof(void*), "Inconsistent sizes!");

void fill_in_parameter_sizes(size_t* param_sizes, size_t* param_sizeof, GPT2Config config) {
    size_t Vp = config.padded_vocab_size;
    size_t C = config.channels;
    size_t maxT = config.max_seq_len;
    size_t L = config.num_layers;
    param_sizes[0] = Vp * C; // wte
    param_sizes[1] = maxT * C; // wpe
    param_sizes[2] = L * C; // ln1w
    param_sizes[3] = L * C; // ln1b
    param_sizes[4] = L * (3 * C) * C; // qkvw
    param_sizes[5] = L * (3 * C); // qkvb
    param_sizes[6] = L * C * C; // attprojw
    param_sizes[7] = L * C; // attprojb
    param_sizes[8] = L * C; // ln2w
    param_sizes[9] = L * C; // ln2b
    param_sizes[10] = L * (4 * C) * C; // fcw
    param_sizes[11] = L * (4 * C); // fcb
    param_sizes[12] = L * C * (4 * C); // fcprojw
    param_sizes[13] = L * C; // fcprojb
    param_sizes[14] = C; // lnfw
    param_sizes[15] = C; // lnfb

    // populate the parameter sizes in bytes (all the same for now, keeping for future use)
    for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        param_sizeof[i] = sizeof(floatX);
    }
}

// allocate memory for the parameters and point the individual tensors to the right places
void* malloc_and_point_parameters(ParameterTensors* params, size_t* param_elements, size_t *param_sizeof) {
    // calculate the total number of parameters and bytes across all tensors
    size_t num_parameters_bytes = 0;
    for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        num_parameters_bytes += param_elements[i] * param_sizeof[i];
    }
    // malloc all parameters all at once on the device
    void* params_memory;
    cudaCheck(hipMalloc((void**)&params_memory, num_parameters_bytes));
    // assign all the tensors their place in the array
    floatX** ptrs[] = {
        &params->wte, &params->wpe, &params->ln1w, &params->ln1b, &params->qkvw, &params->qkvb,
        &params->attprojw, &params->attprojb, &params->ln2w, &params->ln2b, &params->fcw, &params->fcb,
        &params->fcprojw, &params->fcprojb, &params->lnfw, &params->lnfb
    };
    char* params_memory_iterator = (char*)params_memory;
    for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        *(ptrs[i]) = (floatX*)params_memory_iterator;
        params_memory_iterator += param_elements[i] * param_sizeof[i];
    }
    return params_memory;
}

#define NUM_ACTIVATION_TENSORS 21
typedef struct {
    floatX* encoded; // (B, T, C)
    floatX* ln1; // (L, B, T, C)
    floatX* ln1_mean; // (L, B, T)
    floatX* ln1_rstd; // (L, B, T)
    floatX* atty; // (L, B, T, C)
    floatX* att; // (L, B, NH, T, T) (smaller with cuDNN)
    floatX* attproj; // (L, B, T, C)
    floatX* residual2; // (L, B, T, C)
    floatX* ln2; // (L, B, T, C)
    floatX* ln2_mean; // (L, B, T)
    floatX* ln2_rstd; // (L, B, T)
    floatX* fch; // (L, B, T, 4*C)
    floatX* fch_gelu; // (L, B, T, 4*C)
    floatX* fcproj; // (L, B, T, C)
    floatX* residual3; // (L, B, T, C)
    floatX* lnf; // (B, T, C);   if LN recomputation is enabled (-r 2 and above), will be used for _all_ layernorms
    floatX* lnf_mean; // (B, T)
    floatX* lnf_rstd; // (B, T)
    floatX* losses; // (B, T)
    // adding these two compared to the CPU .c code, needed for attention kernel as buffers
    floatX* qkvr; // (L, B, T, 3*C)
    // in inference mode, this buffer will store the logits
    // in training mode, this buffer will contain the *gradients* of the logits.
    // during the processing of transformer blocks, we will also use this as a
    // general scratchpad buffer. Allocation is made large enough to hold (B, T, 3C),
    // (B, NH, T, T), and (B, T, V) shaped tensors.
    floatX* output;
} ActivationTensors;

void fill_in_activation_sizes(size_t* act_sizes, size_t B, size_t T, GPT2Config config, int recompute) {
    size_t Vp = config.padded_vocab_size;
    size_t L = config.num_layers;
    size_t NH = config.num_heads;
    size_t C = config.channels;
    act_sizes[0] = B * T * C; // encoded
    // if recompute >= 1 then we will recompute the layernorm forward activation during backward pass
    act_sizes[1] = (recompute < 2) ? L * B * T * C : 0; // ln1
    act_sizes[2] = L * B * T; // ln1_mean
    act_sizes[3] = L * B * T; // ln1_rstd
    act_sizes[4] = L * B * T * C; // atty
    #ifdef ENABLE_CUDNN
    // FP32 stats tensor for cuDNN to be passed to backward pass
    act_sizes[5] = L * B * NH * T * (sizeof(float) / sizeof(floatX));
    #else
    act_sizes[5] = L * B * NH * T * T; // att
    #endif
    act_sizes[6] = L * B * T * C; // attproj
    act_sizes[7] = L * B * T * C; // residual2
    // if recompute >= 1 then we will recompute the layernorm forward activation during backward pass
    act_sizes[8] = (recompute < 2) ? L * B * T * C : 0; // ln2
    act_sizes[9] = L * B * T; // ln2_mean
    act_sizes[10] = L * B * T; // ln2_rstd
    act_sizes[11] = L * B * T * 4*C; // fch
    // if recompute >= 1 then we will recompute gelu_forward during backward and use this as scratch buffer
    act_sizes[12] = (recompute < 1) ? L * B * T * 4*C : B * T * 4*C;
    act_sizes[13] = L * B * T * C; // fcproj
    act_sizes[14] = L * B * T * C; // residual3
    act_sizes[15] = B * T * C; // lnf
    act_sizes[16] = B * T; // lnf_mean
    act_sizes[17] = B * T; // lnf_rstd
    act_sizes[18] = B * T; // losses
    act_sizes[19] = L * B * T * 3*C; // qkvr
    act_sizes[20] = B * T * max(3*C, max(NH*T, Vp)); // output / scratch
}

// Backward pass is conceptually quite different from forward, because we can discard
// the activations of a layer as soon as we're done with it. This lets us aggressively
// reuse memory, so that we need far fewer tensors for backward state.
#ifdef ENABLE_CUDNN
#define NUM_BACKWARD_TENSORS 2
#else
#define NUM_BACKWARD_TENSORS 3
#endif

typedef struct {
    floatX* bt4c; // (B, T, 4*C)
    floatX* residual3; // (B, T, C)
    #ifndef ENABLE_CUDNN
    floatX* preatt; // (B, NH, T, T)
    #endif
} GradActTensors;

void fill_in_grad_act_sizes(size_t* act_sizes, size_t B, size_t T, GPT2Config config) {
    size_t C = config.channels;
    act_sizes[0] = B * T * 4 * C; // bt4c
    act_sizes[1] = B * T * C; // residual3

    #ifndef ENABLE_CUDNN
    size_t NH = config.num_heads;
    act_sizes[2] = B * NH * T * T; // preatt
    #endif
}

void* malloc_and_point(floatX** targets[], const size_t* act_sizes, size_t n) {
    size_t num_activations = 0;
    for (size_t i = 0; i < n; i++) {
        num_activations += act_sizes[i];
    }
    void* acts_memory;
    cudaCheck(hipMalloc((void**)&acts_memory, num_activations * sizeof(floatX)));
    char* acts_memory_iterator = (char*)acts_memory;
    for (size_t i = 0; i < n; i++) {
        // extra protection so we don't accidentally use an empty buffer
        if(act_sizes[i] == 0) {
            *(targets[i]) = NULL;
        }else {
            *(targets[i]) = (floatX*) acts_memory_iterator;
            acts_memory_iterator += act_sizes[i] * sizeof(floatX);
        }
    }
    return acts_memory;
}

void* malloc_and_point_activations(ActivationTensors* acts, const size_t* act_sizes) {
    floatX** ptrs[] = {
        &acts->encoded, &acts->ln1, &acts->ln1_mean, &acts->ln1_rstd, &acts->atty,
        &acts->att, &acts->attproj, &acts->residual2, &acts->ln2, &acts->ln2_mean,
        &acts->ln2_rstd, &acts->fch, &acts->fch_gelu, &acts->fcproj, &acts->residual3, &acts->lnf,
        &acts->lnf_mean, &acts->lnf_rstd, &acts->losses, &acts->qkvr, &acts->output
    };
    return malloc_and_point(ptrs, act_sizes, NUM_ACTIVATION_TENSORS);
}

void* malloc_and_point_backward(GradActTensors* acts, const size_t* act_sizes) {
    floatX** ptrs[] = {
        &acts->bt4c, &acts->residual3,
        #ifndef ENABLE_CUDNN
        &acts->preatt,
        #endif
    };
    return malloc_and_point(ptrs, act_sizes, NUM_BACKWARD_TENSORS);
}

typedef struct {
    GPT2Config config;
    // the weights of the model, and their sizes
    ParameterTensors params;
    size_t param_elements[NUM_PARAMETER_TENSORS];
    size_t param_sizeof[NUM_PARAMETER_TENSORS];
    void* params_memory;
    size_t num_parameters;
    size_t num_parameters_bytes;
    // gradients of the weights
    ParameterTensors grads;
    void* grads_memory;
    // buffers for the AdamW optimizer
    float* m_memory;
    float* v_memory;
    float* master_weights;     // is NULL unless fp32 weights is enabled.
    // the activations of the model, and their sizes
    ActivationTensors acts;
    size_t act_sizes[NUM_ACTIVATION_TENSORS];
    void* acts_memory;
    size_t num_activations;
    // gradients of the activations
    GradActTensors grads_acts;
    size_t num_grad_acts;
    void* grads_acts_memory;
    // other run state configuration
    int batch_size; // the batch size (B) of current forward pass
    int seq_len; // the sequence length (T) of current forward pass
    int* inputs; // the input tokens for the current forward pass
    int* targets; // the target tokens for the current forward pass
    float mean_loss; // after a forward pass with targets, will be populated with the mean loss
    float accumulated_mean_loss; // Mean loss after aggregating it on all GPUs
    floatX* cpu_losses; // CPU buffer to copy the losses to, allocated with hipHostMalloc
    float* cpu_losses_fp32; // same but fp32
    unsigned long long rng_state; // the RNG state for seeding stochastic rounding etc.
    int use_master_weights; // keep master weights copy in float for optim update? 0|1
    int recompute; // recompute gelu | layernorm forward during model backward? 0|1|2
    // todo - if other functions need cpu scratch buffers in the future, reuse as generic scratch?
    int* workload_indices; // encoder_backward, B*T*num_c_groups (int)
    int4* bucket_info;     // encoder_backward, B*T*num_c_groups (int4) - size for worst case
} GPT2;

void gpt2_init_common(GPT2 *model) {
    // common inits outside of the model weights
    // the weights are initialized either in:
    // - gpt2_build_from_checkpoint() if loading from a checkpoint
    // - gpt2_build_from_random() if starting from scratch
    // memory lazily initialized in forward()
    model->acts_memory = NULL;
    model->inputs = NULL;
    model->targets = NULL;
    model->cpu_losses = NULL;
    model->cpu_losses_fp32 = NULL;
    // the B,T params are determined and set, fixed on first batch in forward()
    model->batch_size = 0;
    model->seq_len = 0;
    model->mean_loss = -1.0f; // -1.0f designates no loss, set at end of forward()
    // memory lazily initialized in backward()
    model->grads_memory = NULL;
    model->grads_acts_memory = NULL;
    model->workload_indices = NULL; // on cpu, for encoder_backward
    model->bucket_info = NULL; // on cpu, for encoder_backward
    // memory lazily initialized in update()
    model->m_memory = NULL;
    model->v_memory = NULL;
    model->master_weights = NULL;
    // other default settings
    model->rng_state = 13371337; // used in stochastic rounding
    model->use_master_weights = 1; // safe default: do keep master weights in fp32
    model->recompute = 1; // good default: recompute gelu but not layernorm
}

void gpt2_write_to_checkpoint(GPT2 *model, const char* checkpoint_path) {
    // write the model to a checkpoint file
    printf0("Writing model to %s\n", checkpoint_path);
    FILE *model_file = fopenCheck(checkpoint_path, "wb");
    // write the header first
    int model_header[256];
    memset(model_header, 0, sizeof(model_header));
    model_header[0] = 20240326; // magic number
    assert(PRECISION_MODE == PRECISION_FP32 || PRECISION_MODE == PRECISION_BF16);
    model_header[1] = PRECISION_MODE == PRECISION_FP32 ? 3 : 5; // version
    model_header[2] = model->config.max_seq_len;
    model_header[3] = model->config.vocab_size;
    model_header[4] = model->config.num_layers;
    model_header[5] = model->config.num_heads;
    model_header[6] = model->config.channels;
    model_header[7] = model->config.padded_vocab_size;
    fwrite(model_header, sizeof(int), 256, model_file);
    // write the parameters
    void* params_memory_cpu = (void*)mallocCheck(model->num_parameters_bytes);
    cudaCheck(hipMemcpy(params_memory_cpu, model->params_memory, model->num_parameters_bytes, hipMemcpyDeviceToHost));
    fwrite(params_memory_cpu, 1, model->num_parameters_bytes, model_file);
    free(params_memory_cpu);
    // close file, we're done
    fcloseCheck(model_file);
}

void gpt2_build_from_checkpoint(GPT2 *model, const char* checkpoint_path) {

    if (PRECISION_MODE == PRECISION_FP16) {
        // TODO for later perhaps, would require us dynamically converting the
        // model weights from fp32 to fp16 online, here in this function, or writing
        // the fp16 weights directly from Python, which we only do for fp32/bf16 atm.
        fprintf(stderr, "build_from_checkpoint() does not support fp16 right now.\n");
        exit(EXIT_FAILURE);
    }

    // read in model from a checkpoint file
    FILE *model_file = fopenCheck(checkpoint_path, "rb");
    int model_header[256];
    freadCheck(model_header, sizeof(int), 256, model_file);
    if (model_header[0] != 20240326) { printf("Bad magic model file\n"); exit(EXIT_FAILURE); }
    int version = model_header[1];
    if (!(version == 3 || version == 5)) {
        // 3 = fp32, padded vocab
        // 5 = bf16, padded vocab, layernorms also in bf16
        fprintf(stderr, "Bad version in model file\n");
        fprintf(stderr, "---> HINT: try to re-run `python train_gpt2.py`\n");
        exit(EXIT_FAILURE);
    }
    if (PRECISION_MODE == PRECISION_BF16 && version != 5) {
        fprintf(stderr, "Precision is configured as BF16 but model at %s is not.\n", checkpoint_path);
        fprintf(stderr, "---> HINT: are you sure you're loading a _bf16.bin file?\n");
        exit(EXIT_FAILURE);
    }
    if (PRECISION_MODE == PRECISION_FP32 && version != 3) {
        fprintf(stderr, "Precision is configured as FP32 but model at %s is not.\n", checkpoint_path);
        fprintf(stderr, "---> HINT: to turn on FP32 you have to compile like: `make train_gpt2cu PRECISION=FP32`\n");
        fprintf(stderr, "---> HINT: are you sure you're loading a .bin file without any _bf16 in the name?\n");
        exit(EXIT_FAILURE);
    }

    // read in hyperparameters
    model->config.max_seq_len = model_header[2];
    model->config.vocab_size = model_header[3];
    model->config.num_layers = model_header[4];
    model->config.num_heads = model_header[5];
    model->config.channels = model_header[6];
    model->config.padded_vocab_size = model_header[7];

    // allocate space for all the parameters and read them in
    fill_in_parameter_sizes(model->param_elements, model->param_sizeof, model->config);

    model->num_parameters = 0;
    model->num_parameters_bytes = 0;
    for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        model->num_parameters += model->param_elements[i];
        model->num_parameters_bytes += model->param_elements[i] * model->param_sizeof[i];
    }

    // create memory for model parameters on the device
    model->params_memory = malloc_and_point_parameters(&model->params, model->param_elements, model->param_sizeof);

    // read in all the parameters from file and copy them to device
    void* params_memory_cpu = (void*)mallocCheck(model->num_parameters_bytes);
    freadCheck(params_memory_cpu, 1, model->num_parameters_bytes, model_file);
    cudaCheck(hipMemcpy(model->params_memory, params_memory_cpu, model->num_parameters_bytes, hipMemcpyHostToDevice));
    free(params_memory_cpu);
    fcloseCheck(model_file);

    gpt2_init_common(model);
    // only return from this function once we are certain the params are ready on the GPU
    cudaCheck(hipDeviceSynchronize());
}

void gpt2_build_from_random(GPT2 *model, int depth) {
    // init random (training from scratch)

    // parameterize the size of gpt2 based only on the depth of the model (num_layers)
    model->config.num_layers = depth;
    // follows GPT-2 sizes
    int channels, num_heads;
    if      (depth == 6)  { channels = 384; num_heads = 6; } // gpt2-tiny (30M)
    else if (depth == 12) { channels = 768; num_heads = 12; } // gpt2 (124M)
    else if (depth == 24) { channels = 1024; num_heads = 16; } // gpt2-medium (350M)
    else if (depth == 36) { channels = 1280; num_heads = 20; } // gpt2-large (774M)
    else if (depth == 48) { channels = 1600; num_heads = 25; } // gpt2-xl (1558M)
    else { fprintf(stderr, "Unsupported depth for now\n"); exit(EXIT_FAILURE); }
    model->config.channels = channels;
    model->config.num_heads = num_heads;
    model->config.max_seq_len = 1024;
    model->config.vocab_size = 50257;
    model->config.padded_vocab_size = 50304; // padded to 128

    // fill in all the parameter tensor dimensions and types
    fill_in_parameter_sizes(model->param_elements, model->param_sizeof, model->config);
    model->num_parameters = 0;
    model->num_parameters_bytes = 0;
    for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        model->num_parameters += model->param_elements[i];
        model->num_parameters_bytes += model->param_elements[i] * model->param_sizeof[i];
    }
    // create memory for model parameters on the device
    model->params_memory = malloc_and_point_parameters(&model->params, model->param_elements, model->param_sizeof);

    // allocate and random init the memory for all the parameters with GPT-2 schema
    // weights ~N(0, 0.02), biases 0, c_proj weights ~N(0, 0.02/(2*L)**0.5)
    // NOTE: assuming all parameters are of the type floatX, could be relaxed later
    mt19937_state init_rng;
    manual_seed(&init_rng, 42);
    floatX* params_memory_cpu = (floatX*)mallocCheck(model->num_parameters_bytes);
    memset(params_memory_cpu, 0, model->num_parameters_bytes);
    // fill in all the weights with random values
    float residual_scale = 1.0f / sqrtf(2.0f * model->config.num_layers);
    // we have to init all these tensors exactly in the order that PyTorch initializes them
    // so that we can match them up and get correctness and exactly the same initial conditions
    size_t L = model->config.num_layers;
    size_t offset = 0;
    for (int l = 0; l < L; l++) {
        offset = 0;
        for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
            // the layernorm parameters are all initialized to 1
            if (l == 0 && (i == 2 || i == 8 || i == 14)) { // only at l = 0 to init these just once
                for (size_t j = 0; j < model->param_elements[i]; j++) {
                    params_memory_cpu[offset + j] = 1.0f;
                }
            }
            // weights tensors are handled here
            if ((l == 0 && (i == 0 || i == 1)) // only at l = 0, init the wte and wpe tensors
              || i == 4 || i == 6 || i == 10 || i == 12) {
                int n = model->param_elements[i];
                size_t layer_offset = 0;
                if (i == 0) {
                    // for wte tensor (padded vocab) override to init V instead of Vp rows
                    n = model->config.vocab_size * model->config.channels;
                }
                if (i == 4 || i == 6 || i == 10 || i == 12) {
                    // weight tensors, we are only initializing layer l
                    assert(n % L == 0);
                    n = n / L;
                    layer_offset = l * n;
                }
                // in GPT-2, the projections back into the residual stream are additionally
                // scaled by 1/sqrt(2*L) for training stability
                float scale = (i == 6 || i == 12) ? 0.02f * residual_scale : 0.02f;
                // okay let's draw the random numbers and write them
                float *fp32_buffer = (float*)mallocCheck(n * sizeof(float));
                normal_(fp32_buffer, n, 0.0f, scale, &init_rng);
                for (size_t j = 0; j < n; j++) {
                    params_memory_cpu[offset + layer_offset + j] = (floatX)fp32_buffer[j];
                }
                free(fp32_buffer);
            }
            offset += model->param_elements[i];
        }
    }

    // copy them to GPU
    cudaCheck(hipMemcpy(model->params_memory, params_memory_cpu, model->num_parameters_bytes, hipMemcpyHostToDevice));
    free(params_memory_cpu);

    gpt2_init_common(model);
}

void gpt2_forward(GPT2 *model, const int* inputs, const int* targets, size_t B, size_t T, int grad_accum_steps=1) {
    // right now, this function is fully synchronous with the host
    NVTX_RANGE_FN();
    // targets are optional and could be NULL
    // in this function we must be careful and use size_t instead of int, otherwise
    // we could overflow int. E.g. l * B * NH * T * T overflows int at B 16.

    // ensure the model was initialized or error out
    if (model->params_memory == NULL) {
        printf("Error: model was not initialized properly.\n");
        exit(EXIT_FAILURE);
    }

    // convenience parameters
    const size_t V = model->config.vocab_size;
    const size_t Vp = model->config.padded_vocab_size;
    const size_t L = model->config.num_layers;
    const size_t NH = model->config.num_heads;
    const size_t C = model->config.channels;

    // allocate space for all the activations if needed (done here, lazily)
    if(model->acts_memory == NULL) {
        NvtxRange rng("InitActs");
        // record the current B,T as well
        model->batch_size = B;
        model->seq_len = T;
        // allocate the space
        fill_in_activation_sizes(model->act_sizes, B, T, model->config, model->recompute);
        size_t num_activations = 0;
        for (size_t i = 0; i < NUM_ACTIVATION_TENSORS; i++) {
            num_activations += model->act_sizes[i];
        }
        model->num_activations = num_activations;
        printf0("allocating %d MiB for activations\n", (int)round(num_activations * sizeof(floatX) / (1024 * 1024)));
        model->acts_memory = malloc_and_point_activations(&model->acts, model->act_sizes);
        // also create memory for caching inputs and targets
        cudaCheck(hipMalloc((void**)&model->inputs, B * T * sizeof(int)));
        cudaCheck(hipMalloc((void**)&model->targets, B * T * sizeof(int)));
        cudaCheck(hipHostMalloc((void**)&model->cpu_losses, B * T * sizeof(floatX)));
        cudaCheck(hipHostMalloc((void**)&model->cpu_losses_fp32, B * T * sizeof(float)));
    } else {
        // validate B,T is consistent with how we've allocated the memory before
        // in principle we could get more clever here in the future, for now this is safest
        if (B != model->batch_size || T != model->seq_len) {
            printf("Model: B=%d T=%d, Desired: B=%d T=%d\n", model->batch_size, model->seq_len, (int)B, (int)T);
            exit(EXIT_FAILURE);
        }
    }

    // copy inputs/targets to the model
    cudaCheck(hipMemcpy(model->inputs, inputs, B * T * sizeof(int), hipMemcpyHostToDevice));
    if (targets != NULL) {
        cudaCheck(hipMemcpy(model->targets, targets, B * T * sizeof(int), hipMemcpyHostToDevice));
    }

    // validate inputs, all indices must be in the range [0, V)
    // we can do this while the copies are already underway
    for(int i = 0; i < B * T; i++) {
        assert(0 <= inputs[i] && inputs[i] < V);
        if (targets != NULL) {
            assert(0 <= targets[i] && targets[i] < V);
        }
    }

    // forward pass
    ParameterTensors params = model->params; // for brevity
    ActivationTensors acts = model->acts;
    encoder_forward(acts.encoded, model->inputs, params.wte, params.wpe, B, T, C, main_stream); // encoding goes into residual[0]

    // first layernorm isn't fused
    layernorm_forward((model->recompute < 2) ? acts.ln1 : acts.lnf, acts.ln1_mean, acts.ln1_rstd, acts.encoded, params.ln1w, params.ln1b, B, T, C, main_stream);

    for (int l = 0; l < L; l++) {
        NvtxRange layer_range("Layer", l);

        floatX* residual = l == 0 ? acts.encoded : acts.residual3 + (l-1) * B * T * C;

        // get the pointers of the weights for this layer
        floatX* l_qkvw = params.qkvw + l * 3*C * C;
        floatX* l_qkvb = params.qkvb + l * 3*C;
        floatX* l_attprojw = params.attprojw + l * C * C;
        floatX* l_attprojb = params.attprojb + l * C;
        floatX* l_ln2w = params.ln2w + l * C;
        floatX* l_ln2b = params.ln2b + l * C;
        floatX* l_fcw = params.fcw + l * 4*C * C;
        floatX* l_fcb = params.fcb + l * 4*C;
        floatX* l_fcprojw = params.fcprojw + l * C * 4*C;
        floatX* l_fcprojb = params.fcprojb + l * C;

        // get the pointers of the activations for this layer
        floatX* l_ln1 = (model->recompute < 2) ? acts.ln1 + l * B * T * C : acts.lnf;
        floatX* l_qkvr = acts.qkvr + l * B * T * 3*C;
        floatX* l_atty = acts.atty + l * B * T * C;
        floatX* l_attproj = acts.attproj + l * B * T * C;
        floatX* l_residual2 = acts.residual2 + l * B * T * C;
        floatX* l_ln2 = (model->recompute < 2) ? acts.ln2 + l * B * T * C : acts.lnf;
        floatX* l_ln2_mean = acts.ln2_mean + l * B * T;
        floatX* l_ln2_rstd = acts.ln2_rstd + l * B * T;
        floatX* l_fch = acts.fch + l * B * T * 4*C;
        // reuse the same activation buffer at each layer, as we'll re-compute the gelu during backward
        // very useful because we dramatically reduce VRAM usage, and may be able to fit larger batch size
        floatX* l_fch_gelu = (model->recompute < 1) ? acts.fch_gelu + l * B * T * 4*C : acts.fch_gelu;
        floatX* l_fcproj = acts.fcproj + l * B * T * C;
        floatX* l_residual3 = acts.residual3 + l * B * T * C;

        // now do the forward pass
        #ifdef ENABLE_CUDNN
        float* l_att = (float*)acts.att + l * B * NH * T; // cuDNN needs a smaller FP32 tensor
        matmul_forward_cublaslt(l_qkvr, l_ln1, l_qkvw, l_qkvb, B, T, C, 3*C, main_stream);
        attention_forward_cudnn(l_atty, (float*)l_att, l_qkvr, B, T, NH, C, main_stream);
        #else
        floatX* l_att = acts.att + l * B * NH * T * T;
        // these are only needed as scratchpads for the forward pass, but
        // need not be stored for backward
        floatX* scratch = (floatX*)acts.output;
        matmul_forward_cublaslt(scratch, l_ln1, l_qkvw, l_qkvb, B, T, C, 3*C, main_stream);
        attention_forward(l_atty, l_qkvr, l_att, scratch, B, T, C, NH, main_stream);
        #endif

        matmul_forward_cublaslt(l_attproj, l_atty, l_attprojw, l_attprojb, B, T, C, C, main_stream);
        fused_residual_forward5(l_residual2, l_ln2, l_ln2_mean, l_ln2_rstd, residual, l_attproj, l_ln2w, l_ln2b, B*T, C, main_stream);
        matmul_forward_cublaslt(l_fch, l_ln2, l_fcw, l_fcb, B, T, C, 4*C, main_stream);
        gelu_forward(l_fch_gelu, l_fch, B*T*4*C, main_stream);
        matmul_forward_cublaslt(l_fcproj, l_fch_gelu, l_fcprojw, l_fcprojb, B, T, 4*C, C, main_stream);

        // OK, fusion across blocks.
        if(l+1 != L) {
            floatX* l_ln1 = (model->recompute < 2) ? acts.ln1 + (l + 1) * B * T * C : acts.lnf;
            floatX* l_ln1_mean = acts.ln1_mean + (l + 1) * B * T;
            floatX* l_ln1_rstd = acts.ln1_rstd + (l + 1) * B * T;
            const floatX* l_ln1w = params.ln1w + (l + 1) * C;
            const floatX* l_ln1b = params.ln1b + (l + 1) * C;
            fused_residual_forward5(l_residual3, l_ln1, l_ln1_mean, l_ln1_rstd, l_residual2, l_fcproj, l_ln1w, l_ln1b,
                                    B * T, C, main_stream);
        } else {
            fused_residual_forward5(l_residual3, acts.lnf, acts.lnf_mean, acts.lnf_rstd, l_residual2, l_fcproj,
                                    params.lnfw, params.lnfb,
                                    B * T, C, main_stream);
        }
    }

    matmul_forward_cublaslt(acts.output, acts.lnf, params.wte, NULL, B, T, C, Vp, main_stream);

    // also forward the cross-entropy loss function if we have the targets
    if (targets != NULL) {
        NvtxRange classifier_and_loss_range("classifier_and_loss");
        // fused classifier: does the forward pass and first part of the backward pass
        const float dloss = 1.0f / (B * T * grad_accum_steps); // results in the uniform average loss over all elements
        fused_classifier(acts.output, acts.losses, dloss, model->targets, B, T, V, Vp, main_stream);
        // for convenience also evaluate the mean loss (TODO re-think this compute+sync point)
        cudaCheck(hipMemcpy(model->cpu_losses, acts.losses, B * T * sizeof(floatX), hipMemcpyDeviceToHost));
        float mean_loss = 0.0f;
        for (int i = 0; i < B*T; i++) {
            float loss = (float)(model->cpu_losses[i]);
            model->cpu_losses_fp32[i] = loss;
            mean_loss += loss;
        }
        mean_loss /= B*T*grad_accum_steps;
        model->mean_loss = mean_loss;
    } else {
        // if we don't have targets, we don't have loss
        model->mean_loss = -1.0f;
    }
    cudaCheck(hipDeviceSynchronize());
}

void gpt2_zero_grad(GPT2 *model) {
    NVTX_RANGE_FN();
    if (model->grads_memory != NULL) {
        cudaCheck(hipMemset(model->grads_memory, 0, model->num_parameters * sizeof(floatX)));
    }
    cudaCheck(hipDeviceSynchronize());
}

void gpt2_backward(GPT2 *model, int* inputs) {
    NVTX_RANGE_FN();
    // double check we forwarded previously, with targets
    if (model->mean_loss == -1.0f) {
        printf("Error: must forward with targets before backward\n");
        exit(EXIT_FAILURE);
    }

    // lazily allocate the memory for gradients of the weights and activations, if needed
    if (model->grads_memory == NULL) {
        NvtxRange rng("InitGrads");
        // allocate buffers for weight gradients
        printf0("allocating %d MiB for parameter gradients\n", (int)round(model->num_parameters * sizeof(floatX) / (1024 * 1024)));
        model->grads_memory = malloc_and_point_parameters(&model->grads, model->param_elements, model->param_sizeof);
        // we're going to be clever for the activations backward pass. we don't need to exactly
        // mirror the forward pass activations and we will save memory.
        size_t bw_act_sizes[NUM_BACKWARD_TENSORS];
        fill_in_grad_act_sizes(bw_act_sizes, model->batch_size, model->seq_len, model->config);
        // count up and allocate the space
        model->num_grad_acts = 0;
        for (size_t i = 0; i < NUM_BACKWARD_TENSORS; i++) {
            model->num_grad_acts += bw_act_sizes[i];
        }
        printf0("allocating %d MiB for activation gradients\n", (int)round(model->num_grad_acts * sizeof(floatX) / (1024 * 1024)));
        model->grads_acts_memory = malloc_and_point_backward(&model->grads_acts, bw_act_sizes);
        // init gradients of parameters and activations to zero
        gpt2_zero_grad(model);
        // initialise cpu scratch buffers for encoder backward
        size_t num_c_groups = CEIL_DIV(model->config.channels, (WARP_SIZE * x128::size));
        assert((size_t)(model->batch_size * model->seq_len) * num_c_groups < (1ULL<<31ULL)); // todo - maybe an issue for llama3-400B(?)
        model->workload_indices = (int*)mallocCheck(sizeof(int) * model->batch_size * model->seq_len * num_c_groups);
        model->bucket_info = (int4*)mallocCheck(sizeof(int4) * model->batch_size * model->seq_len * num_c_groups);
    }

    // convenience shortcuts, size_t instead of int so that pointer arithmetics don't overflow
    const size_t B = model->batch_size;
    const size_t T = model->seq_len;
    const size_t Vp = model->config.padded_vocab_size;
    const size_t L = model->config.num_layers;
    const size_t NH = model->config.num_heads;
    const size_t C = model->config.channels;

    // backward pass: go in the reverse order of the forward pass, and call backward() functions
    ParameterTensors params = model->params; // for brevity
    ParameterTensors grads = model->grads;
    ActivationTensors acts = model->acts;
    GradActTensors grads_acts = model->grads_acts;

    // reset residual stream gradients (put here to work with gradient accumulation)
    cudaCheck(hipMemset(model->grads_acts.residual3, 0, B * T * C * sizeof(floatX)));

    // re-use the output buffer of the forward pass as a scratchpad during backward pass
    float*  scratchF = (float*)acts.output;
    floatX* scratchX = (floatX*)acts.output;

    // we kick off the chain rule by filling in dlosses with 1.0f/(B*T)
    // this was done in the fused classifier kernel as last step of forward pass
    // technically that is a small, inline backward() pass of calculating
    // total, final loss as the mean over all losses over all (B,T) positions in the batch
    // next: backward the classifier matmul
    matmul_backward(grads_acts.bt4c, grads.wte, NULL, acts.output, acts.lnf, params.wte, NULL, B, T, C, Vp, main_stream);
    // backward the final layernorm
    floatX* residual = acts.residual3 + (L-1) * B * T * C; // last residual is in residual3
    floatX* dresidual = (floatX*)grads_acts.residual3; // the main buffer holding the gradient in the backward pass
    layernorm_backward(dresidual, grads.lnfw, grads.lnfb, scratchF, grads_acts.bt4c, residual, params.lnfw, acts.lnf_mean, acts.lnf_rstd, B, T, C, main_stream);

    // from this point on, we no longer need the values stored in the last residual, so we can reuse that memory as generic
    // scratch for backward computations
    floatX* dl_btc = residual;

    // now backward all the layers
    for (int l = L-1; l >= 0; l--) {
        NvtxRange layer_range("Layer", l);

        residual = l == 0 ? acts.encoded : acts.residual3 + (l-1) * B * T * C;

        // get the pointers of the weights for this layer
        floatX* l_ln1w = params.ln1w + l * C;
        floatX* l_ln1b = params.ln1b + l * C;
        floatX* l_qkvw = params.qkvw + l * 3*C * C;
        floatX* l_attprojw = params.attprojw + l * C * C;
        floatX* l_ln2w = params.ln2w + l * C;
        floatX* l_ln2b = params.ln2b + l * C;
        floatX* l_fcw = params.fcw + l * 4*C * C;
        floatX* l_fcprojw = params.fcprojw + l * C * 4*C;
        // get the pointers of the gradients of the weights for this layer
        floatX* dl_ln1w = grads.ln1w + l * C;
        floatX* dl_ln1b = grads.ln1b + l * C;
        floatX* dl_qkvw = grads.qkvw + l * 3*C * C;
        floatX* dl_qkvb = grads.qkvb + l * 3*C;
        floatX* dl_attprojw = grads.attprojw + l * C * C;
        floatX* dl_attprojb = grads.attprojb + l * C;
        floatX* dl_ln2w = grads.ln2w + l * C;
        floatX* dl_ln2b = grads.ln2b + l * C;
        floatX* dl_fcw = grads.fcw + l * 4*C * C;
        floatX* dl_fcb = grads.fcb + l * 4*C;
        floatX* dl_fcprojw = grads.fcprojw + l * C * 4*C;
        floatX* dl_fcprojb = grads.fcprojb + l * C;
        // get the pointers of the activations for this layer
        floatX* l_ln1 = (model->recompute < 2) ? acts.ln1 + l * B * T * C : acts.lnf;
        floatX* l_ln1_mean = acts.ln1_mean + l * B * T;
        floatX* l_ln1_rstd = acts.ln1_rstd + l * B * T;
        floatX* l_qkvr = acts.qkvr + l * B * T * 3*C;
        floatX* l_atty = acts.atty + l * B * T * C;
        floatX* l_residual2 = acts.residual2 + l * B * T * C;
        floatX* l_ln2 = (model->recompute < 2) ? acts.ln2 + l * B * T * C : acts.lnf;
        floatX* l_ln2_mean = acts.ln2_mean + l * B * T;
        floatX* l_ln2_rstd = acts.ln2_rstd + l * B * T;
        floatX* l_fch = acts.fch + l * B * T * 4*C;
        floatX* l_fch_gelu = (model->recompute < 1) ? acts.fch_gelu + l * B * T * 4*C : acts.fch_gelu;
        // get the pointers of the gradients of the activations for this layer
        // notice that there is no l *, because we just have a single copy, and keep
        // re-using this memory in every Transformer block as we calculate backward pass

        floatX* dl_bt4c = (floatX*)grads_acts.bt4c;

        // start the backward pass for this layer
        if(model->recompute >= 1) {
            // recompute >= 1 means we recompute gelu. in this case,
            // l_fch_gelu is just a buffer, so re-compute the gelu from l_fch here
            gelu_forward(l_fch_gelu, l_fch, B*T*4*C, main_stream);
        }
        matmul_backward(dl_bt4c, dl_fcprojw, dl_fcprojb, dresidual, l_fch_gelu, l_fcprojw, scratchF, B, T, 4*C, C, main_stream);
        gelu_backward_inplace(dl_bt4c, l_fch, B*T*4*C, main_stream);
        if(model->recompute >= 2) {
            // same as gelu above, l_ln1 and l_ln2 are just buffers if recompute >= 2, recompute them here on demand
            layernorm_forward(l_ln2, l_ln2_mean, l_ln2_rstd, l_residual2, l_ln2w, l_ln2b, B, T, C, main_stream);
        }
        matmul_backward(dl_btc, dl_fcw, dl_fcb, dl_bt4c, l_ln2, l_fcw, scratchF, B, T, C, 4 * C, main_stream);
        // layernorm backward does += to the dresidual, so it correctly accumulates grad from the MLP block above
        layernorm_backward(dresidual, dl_ln2w, dl_ln2b, scratchF, dl_btc, l_residual2, l_ln2w, l_ln2_mean, l_ln2_rstd, B, T, C, main_stream);
        matmul_backward(dl_btc, dl_attprojw, dl_attprojb, dresidual, l_atty, l_attprojw, scratchF, B, T, C, C, main_stream);

        #ifdef ENABLE_CUDNN
        float* l_att = (float*)acts.att + l * B * NH * T; // cuDNN needs a smaller FP32 tensor
        attention_backward_cudnn(dl_bt4c, dl_btc, l_qkvr, l_atty, (float*)l_att, B, T, NH, C, main_stream);
        #else
        floatX* l_att = acts.att + l * B * NH * T * T;
        // we need B x T x (4)C buffers. l_atty and l_fch aren't needed anymore at this point, so reuse their memory
        floatX* buffer_a = l_atty;
        floatX* buffer_b = l_fch;        // this is B x T x 4C, so even larger than what we need
        floatX* dl_preatt = (floatX*)grads_acts.preatt; // dedicated scratchpad allocation
        attention_backward(dl_bt4c, buffer_b, dl_preatt, scratchX, buffer_a, dl_btc, l_qkvr, l_att, B, T, C, NH, main_stream);
        #endif
        if(model->recompute >= 2) {
            layernorm_forward(l_ln1, l_ln1_mean, l_ln1_rstd, residual, l_ln1w, l_ln1b, B, T, C, main_stream);
        }
        // QKV parameter gradients
        matmul_backward(dl_btc, dl_qkvw, dl_qkvb, dl_bt4c, l_ln1, l_qkvw, scratchF, B, T, C, 3 * C, main_stream);
        // layernorm backward does += to dresidual, so it correctly accumulates gradient for the Attention block above
        layernorm_backward(dresidual, dl_ln1w, dl_ln1b, scratchF, dl_btc, residual, l_ln1w, l_ln1_mean, l_ln1_rstd, B, T, C, main_stream);
    }
    encoder_backward(grads.wte, grads.wpe, scratchX, model->workload_indices, model->bucket_info,
                     dresidual, model->inputs, inputs, B, T, C, random_u32(&model->rng_state), main_stream);

    cudaCheck(hipDeviceSynchronize());
}

// Compute sum of a single CPU value across all GPU processes. No-op when multi-GPU is disabled.
float multi_gpu_cpu_float_sum(float value) {
#ifdef MULTI_GPU
    // note MPI doesn't support all reduce with mean, only sum
    float result;
    mpiCheck(MPI_Allreduce(&value, &result, 1, MPI_FLOAT, MPI_SUM, MPI_COMM_WORLD));
    return result;
#else
    return value;
#endif
}

// Averages out the loss and gradients across all GPUs. No-op when multi-GPU is disabled.
// todo - this version only works if all the parameters are the same size (floatX)
void gpt2_multi_gpu_loss_and_grad_reduce(GPT2* model, MultiGpuConfig* multi_gpu_config) {
#ifdef MULTI_GPU
    NVTX_RANGE_FN();
    // If there's only one process, there is nothing to do
    if (multi_gpu_config->num_processes == 1) { return; }
    // Average all losses.
    model->accumulated_mean_loss = multi_gpu_cpu_float_sum(model->mean_loss) / multi_gpu_config->num_processes;
    // Now average the gradients
    if(multi_gpu_config->zero_stage == 0) {
        // no ZERO == standard DDP: Average all gradients.
        ncclCheck(ncclAllReduce(model->grads_memory, model->grads_memory,
                                model->num_parameters,
                                ncclFloatX, ncclAvg,
                                multi_gpu_config->nccl_comm, main_stream));
    } else if (multi_gpu_config->zero_stage == 1) {
        // ZERO-1: Get the average gradient only for local shard
        floatX* local_grads_memory = (floatX*) model->grads_memory + multi_gpu_config->shard_offset;
        ncclCheck(ncclReduceScatter(model->grads_memory, local_grads_memory,
                                    multi_gpu_config->shard_num_parameters,
                                    ncclFloatX, ncclAvg,
                                    multi_gpu_config->nccl_comm, main_stream));
    }
#endif
    cudaCheck(hipDeviceSynchronize());
}

float gpt2_update(GPT2 *model, float learning_rate, float beta1, float beta2, float eps, float weight_decay, float grad_clip, int t, MultiGpuConfig* multi_gpu_config) {
    // update the model parameters using the AdamW optimizer
    // keep in mind that optimizer sharding (ZeRO-1) assigns different parameters to different GPUs
    // so we may not be responsible for the entire parameter tensor
    // also, this function was very simple a while back but become very complex, only because we want to
    // selectively weight decay some, but not all tensors :(
    // TODO: revisit and probably refactor this entire function
    NVTX_RANGE_FN();
    size_t shard_num_parameters = multi_gpu_config->shard_num_parameters; // num parameters we are responsible for
    size_t shard_offset = multi_gpu_config->shard_offset; // offset into the full parameter tensor
    floatX* params_memory = (floatX*)model->params_memory;
    floatX* grads_memory = (floatX*)model->grads_memory;

    // lazily allocate m,v memory and master weights (usually on the first iteration)
    if (model->m_memory == NULL) {
        NvtxRange rng("InitOpt");
        printf0("allocating %zu MiB for AdamW optimizer state m\n", (shard_num_parameters * sizeof(float)) >> 20);
        printf0("allocating %zu MiB for AdamW optimizer state v\n", (shard_num_parameters * sizeof(float)) >> 20);
        cudaCheck(hipMalloc((void**)&model->m_memory, shard_num_parameters * sizeof(float)));
        cudaCheck(hipMalloc((void**)&model->v_memory, shard_num_parameters * sizeof(float)));
        cudaCheck(hipMemset(model->m_memory, 0, shard_num_parameters * sizeof(float)));
        cudaCheck(hipMemset(model->v_memory, 0, shard_num_parameters * sizeof(float)));
    }
    if (model->use_master_weights == 1 && model->master_weights == NULL) {
        printf0("allocating %zu MiB for master copy of params\n", (shard_num_parameters * sizeof(float)) >> 20);
        cudaCheck(hipMalloc((void**)&model->master_weights, shard_num_parameters * sizeof(float)));
        size_t grid_size = CEIL_DIV(shard_num_parameters, 512);
        copy_and_cast_kernel<<<grid_size, 512, 0, main_stream>>>(model->master_weights, params_memory + shard_offset, shard_num_parameters);
        cudaCheck(hipGetLastError());
    }

    // gradient clipping
    // repurposing this buffer (which isn't needed now) to write grad norm into it
    float* grad_norm_squared = (float*)model->acts.output;
    if (multi_gpu_config->zero_stage == 1) {
        // ^1 because of the ncclReduceScatter() in gpt2_multi_gpu_loss_and_grad_reduce,
        // grads_memory only contains the averaged gradients at the local shard
        // so we only calculate the grad norm at the grads_memory belonging to the local shard
        global_norm_squared(grad_norm_squared, grads_memory + shard_offset, shard_num_parameters, main_stream);
    } else {
        // the ncclAllReduce() in gpt2_multi_gpu_loss_and_grad_reduce has averaged the gradients across all GPUs
        // so each GPU can compute the squared norm over the whole grad vector, with no added comms needed
        global_norm_squared(grad_norm_squared, grads_memory, model->num_parameters, main_stream);
    }
    // transfer the gradient norm to CPU
    float grad_norm_squared_cpu = 0.0f;
    cudaCheck(hipMemcpy(&grad_norm_squared_cpu, grad_norm_squared, sizeof(float), hipMemcpyDeviceToHost));
    if (multi_gpu_config->zero_stage == 1) {
        // further sum the (partial) squared norm across all GPUs (see comment ^1 above)
        grad_norm_squared_cpu = multi_gpu_cpu_float_sum(grad_norm_squared_cpu);
    }

    if(!isfinite(grad_norm_squared_cpu)) {
        // may happen due to some issue (e.g. overflow?)
        // TODO: later may want to keep a global counter of instabilities like this
        printf0("[WARNING]: grad norm is not finite, skipping AdamW update\n");
        return -1.0f;
    }
    float grad_norm_cpu = sqrtf(grad_norm_squared_cpu);
    float grad_scale = (grad_norm_cpu > grad_clip) ? grad_clip / grad_norm_cpu : 1.0f;

    // AdamW update
    unsigned int seed = random_u32(&model->rng_state);
    // individually call the adamw_kernel3 on all parameter tensors separately
    size_t offset = 0;
    for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        size_t num_parameters = model->param_elements[i];
        // the scope of this GPU's work is the range: [shard_offset, shard_offset + shard_num_parameters)
        // this parameter's values are in the range:  [offset, offset + num_parameters)
        // so we are responsible for some of its parameters if:
        // 1) this parameter ends after we begin (i.e. offset + num_parameters > shard_offset)
        // 2) this parameter begins before we end (i.e. offset < shard_offset + shard_num_parameters)
        if(offset + num_parameters > shard_offset && offset < shard_offset + shard_num_parameters) {

            // ok this tensor has at least one element inside the range of responsibility of this GPU
            // let's figure out the exact span we wish to call the AdamW kernel on
            floatX* params_ptr = NULL;
            floatX* grad_ptr = NULL;
            float* m_ptr = NULL;
            float* v_ptr = NULL;
            float* master_ptr = NULL;
            size_t local_params = 0;
            // does the tensor begin before our responsibility?
            if(offset <= shard_offset) {
                // if so, our start point is exactly that of our responsibility, i.e. shard_offset
                params_ptr = params_memory + shard_offset;
                grad_ptr = grads_memory + shard_offset;
                // note that (master_weights, m, v) are already only the "local slice" for this GPU,
                // and are of size shard_num_parameters, instead of the total number of parameters
                // so they do not get offset, i.e. we just start at their index 0
                if (model->master_weights != NULL) { master_ptr = model->master_weights; }
                m_ptr = model->m_memory;
                v_ptr = model->v_memory;
                // the number of parameters we have to update is the minimum of two ranges
                local_params = min(shard_num_parameters, (offset + num_parameters) - shard_offset);
            } else {
                // our start point is the location of this tensor, i.e. offset
                params_ptr = params_memory + offset;
                grad_ptr = grads_memory + offset;
                // this arithmetic gave me a headache but my little doodle example says it's right
                size_t delta = offset - shard_offset;
                if (model->master_weights != NULL) { master_ptr = model->master_weights + delta; }
                m_ptr = model->m_memory + delta;
                v_ptr = model->v_memory + delta;
                local_params = min(num_parameters, shard_num_parameters - delta);
            }
            // we only want to weight decay the 2D tensors and leave all 1D tensors alone
            // in particular this also decays the embedding weights, but this is ok:
            // - the token embeddings are weight shared and participate in the final projection to logits
            // - the position embeddings actively participate at every forward/backward pass
            float wd = (i == 0 || i == 1 || i == 4 || i == 6 || i == 10 || i == 12) ? weight_decay : 0.0f;
            // ok finally call the kernel
            adamw_update(params_ptr, master_ptr, grad_ptr,
                         m_ptr, v_ptr, local_params, learning_rate,
                         beta1, beta2, t, eps, wd, grad_scale, seed, main_stream);
        }
        // advance the offset pointer to the next parameter tensor
        offset += num_parameters;
    }
    cudaCheck(hipGetLastError());

    cudaCheck(hipDeviceSynchronize());
    return grad_norm_cpu;
}

void gpt2_multi_gpu_param_gather(GPT2 *model, MultiGpuConfig* multi_gpu_config)
{
#ifdef MULTI_GPU
    if (multi_gpu_config->num_processes == 1) { return; } // 1 process => noop
    if (multi_gpu_config->zero_stage == 1) {
        // gather updated shards of model->params_memory from each process
        ncclCheck(ncclAllGather((floatX*)model->params_memory + multi_gpu_config->shard_offset, (floatX*)model->params_memory,
                                multi_gpu_config->shard_num_parameters, ncclFloatX,
                                multi_gpu_config->nccl_comm, main_stream));
    }
    cudaCheck(hipGetLastError());
#endif
    cudaCheck(hipDeviceSynchronize());
}

float gpt2_estimate_mfu(GPT2 *model, int num_tokens, float dt) {
    /*
    Estimate model flops utilization (MFU)
    ref: Section 2.1 of https://arxiv.org/pdf/2001.08361
    Note: Ideally, the N here would be only the parameters that actually
    participate in matrix multiplications. In this N, we are over-estimating by
    including LayerNorm params, biases, and the position embedding weights,
    but these are very small terms. Also keep in mind that we would want to exclude
    the token embedding weights, but in GPT-2 these are weight shared, so they
    participate in the classifier matmul, so they are correct to be included in N.
    Note 2: The first term (6 * N) in flops_per_token is all weight matmuls, the
    second is the attention matmul, which is also usually a small contribution.
    */
    size_t N = model->num_parameters;
    int L = model->config.num_layers;
    int C = model->config.channels;
    int T = model->seq_len;
    size_t flops_per_token = 6 * N + (size_t)6 * L * C * T;
    size_t flops_per_step = flops_per_token * num_tokens;
    // express our flops throughput as ratio of A100 bfloat16 peak flops
    float flops_achieved = (float)flops_per_step * (1.0f / dt); // per second
    float flops_promised = get_flops_promised(deviceProp.name, PRECISION_MODE) * 1e12f;
    if(flops_promised < 0) {
        return -1.f;   // don't know
    }
    float mfu = flops_achieved / flops_promised;
    return mfu;
}

void gpt2_free(GPT2 *model) {
    cudaCheck(hipFree(model->params_memory));
    cudaCheck(hipFree(model->grads_memory));
    cudaCheck(hipFree(model->m_memory));
    cudaCheck(hipFree(model->v_memory));
    cudaCheck(hipFree(model->master_weights));
    cudaCheck(hipFree(model->acts_memory));
    cudaCheck(hipFree(model->grads_acts_memory));
    cudaCheck(hipFree(model->inputs));
    cudaCheck(hipFree(model->targets));
    cudaCheck(hipHostFree(model->cpu_losses));
    cudaCheck(hipHostFree(model->cpu_losses_fp32));
    free(model->workload_indices);
    free(model->bucket_info);
}

// ----------------------------------------------------------------------------
// common init & free code for all of train/test/profile

void common_start(bool override_enable_tf32 = true, bool print_device_info = true) {

    // get CUDA device infos
    hipGetDeviceProperties(&deviceProp, multi_gpu_config.local_device_idx);
    if (print_device_info) {
        printf("[System]\n");
        printf("Device %d: %s\n", multi_gpu_config.local_device_idx, deviceProp.name);
    }

    // set up the cuda streams. atm everything is on the single main stream
    hipStreamCreate(&main_stream);
    nvtxNameCudaStreamA(main_stream, "main stream");

    // set up cuBLAS and cuBLASLt
    cublasCheck(hipblasCreate(&cublas_handle));
    cublasCheck(hipblasLtCreate(&cublaslt_handle));
    cudaCheck(hipMalloc(&cublaslt_workspace, cublaslt_workspace_size));

    // TF32 precision is equivalent to torch.set_float32_matmul_precision('high')
    bool enable_tf32 = PRECISION_MODE == PRECISION_FP32 && deviceProp.major >= 8 && override_enable_tf32;
    cublasCheck(hipblasSetMathMode(cublas_handle, enable_tf32 ? HIPBLAS_TF32_TENSOR_OP_MATH : HIPBLAS_DEFAULT_MATH));
    cublas_compute = enable_tf32 ? HIPBLAS_COMPUTE_32F_FAST_TF32 : HIPBLAS_COMPUTE_32F;

    #ifdef ENABLE_CUDNN
    create_cudnn();
    #endif
}

void common_free(GPT2 &model) {
    cudaCheck(hipStreamDestroy(main_stream));
    cudaCheck(hipFree(cublaslt_workspace));
    cublasCheck(hipblasDestroy(cublas_handle));
    cublasCheck(hipblasLtDestroy(cublaslt_handle));
    #ifdef ENABLE_CUDNN
    destroy_cudnn();
    #endif
}

#ifndef TESTING
// if we are TESTING (see test_gpt2.cu), we'll skip everything below this point

// ----------------------------------------------------------------------------
// training resumption logic, very useful when jobs crash once in a while
// the goal is that we can resume optimization from any checkpoint, bit-perfect
// note that "state" refers to things not already saved in the model checkpoint file

void save_state(const char* filename, int step, GPT2* model, DataLoader* loader) {
    printf("Writing state to %s\n", filename);
    FILE *state_file = fopenCheck(filename, "wb");
    int state_header[256];
    memset(state_header, 0, sizeof(state_header));
    // basic identifying information
    state_header[0] = 20240527; // magic number
    state_header[1] = 1; // version number
    state_header[2] = multi_gpu_config.num_processes; // number of processes
    state_header[3] = multi_gpu_config.process_rank; // rank of this process
    // int main state, start at 10 to leave some padding
    state_header[10] = step; // step of the optimization
    // model state, state, start at 20 to leave some padding
    *((unsigned long long*)&state_header[20]) = model->rng_state; // random number generator state
    // dataloader state, start at 30 to leave some padding
    state_header[30] = loader->current_shard; // shard of the dataset
    *((int64_t*)&state_header[31]) = loader->current_position; // position in shard
    fwrite(state_header, sizeof(int), 256, state_file);
    // write AdamW m, v, and master_weights here (they are all float)
    size_t shard_num_parameters = multi_gpu_config.shard_num_parameters;
    float* cpu_buffer = (float*)mallocCheck(shard_num_parameters * sizeof(float));
    cudaCheck(hipMemcpy(cpu_buffer, model->m_memory, shard_num_parameters * sizeof(float), hipMemcpyDeviceToHost));
    fwrite(cpu_buffer, sizeof(float), shard_num_parameters, state_file);
    cudaCheck(hipMemcpy(cpu_buffer, model->v_memory, shard_num_parameters * sizeof(float), hipMemcpyDeviceToHost));
    fwrite(cpu_buffer, sizeof(float), shard_num_parameters, state_file);
    free(cpu_buffer);
    fclose(state_file);
}

void load_state(int* step, GPT2* model, DataLoader* loader, const char* filename) {
    FILE *state_file = fopenCheck(filename, "rb");
    int state_header[256];
    freadCheck(state_header, sizeof(int), 256, state_file);
    assert(state_header[0] == 20240527); // magic number
    assert(state_header[1] == 1); // version number
    assert(state_header[2] == multi_gpu_config.num_processes); // number of processes
    assert(state_header[3] == multi_gpu_config.process_rank); // rank of this process
    *step = state_header[10]; // step of the optimization
    model->rng_state = *((unsigned long long*)&state_header[20]); // random number generator state
    int current_shard = state_header[30]; // shard of the dataset
    int64_t current_position = *((int64_t*)&state_header[31]); // position in shard
    dataloader_resume(loader, current_shard, current_position);
    // read AdamW m, v (they are all float)
    // also allocate the m, v memory in the model, if it does not yet exist
    size_t shard_num_parameters = multi_gpu_config.shard_num_parameters;
    if (model->m_memory == NULL) {
        printf0("allocating %zu MiB for AdamW optimizer state m\n", (shard_num_parameters * sizeof(float)) >> 20);
        printf0("allocating %zu MiB for AdamW optimizer state v\n", (shard_num_parameters * sizeof(float)) >> 20);
        cudaCheck(hipMalloc((void**)&model->m_memory, shard_num_parameters * sizeof(float)));
        cudaCheck(hipMalloc((void**)&model->v_memory, shard_num_parameters * sizeof(float)));
    }
    float* cpu_buffer = (float*)mallocCheck(shard_num_parameters * sizeof(float));
    freadCheck(cpu_buffer, sizeof(float), shard_num_parameters, state_file);
    cudaCheck(hipMemcpy(model->m_memory, cpu_buffer, shard_num_parameters * sizeof(float), hipMemcpyHostToDevice));
    freadCheck(cpu_buffer, sizeof(float), shard_num_parameters, state_file);
    cudaCheck(hipMemcpy(model->v_memory, cpu_buffer, shard_num_parameters * sizeof(float), hipMemcpyHostToDevice));
    free(cpu_buffer);
    fclose(state_file);
}

// ----------------------------------------------------------------------------
// CLI, poor man's argparse
// unclaimed flags lol: k,p

void error_usage() {
    fprintf(stderr, "Usage:   ./train_gpt2cu [options]\n");
    fprintf(stderr, "Options:\n");
    // file system input / output
    fprintf(stderr, "  -i <string> train data filename pattern (default = dev/data/tinyshakespeare/tiny_shakespeare_train.bin)\n");
    fprintf(stderr, "  -j <string> val data filename pattern (default = dev/data/tinyshakespeare/tiny_shakespeare_val.bin)\n");
    fprintf(stderr, "  -e <string> input from model at this filename (default = gpt2_124M_bf16.bin)\n");
    fprintf(stderr, "  -o <string> output log dir (default = NULL, no logging)\n");
    fprintf(stderr, "  -n <int>    write optimization checkpoints every how many steps? (default 0, don't)\n");
    fprintf(stderr, "  -y <int>    resume optimization found inside output log dir? (0=restart/overwrite, 1=resume/append)\n");
    // token layout for each step of the optimization
    fprintf(stderr, "  -b <int>    (per-GPU, micro) batch size B (default = 4)\n");
    fprintf(stderr, "  -t <int>    sequence length T (default = 1024)\n");
    fprintf(stderr, "  -d <int>    total desired batch size (default = B * T * num_processes, i.e. no grad accumulation\n");
    // workload (number of steps)
    fprintf(stderr, "  -x <int>    max_steps of optimization to run (-1 (default) = disable, run 1 epoch)\n");
    // optimization
    fprintf(stderr, "  -l <float>  learning rate (default = 3e-4f)\n");
    fprintf(stderr, "  -u <int>    learning rate warmup iterations (default = 0, no warmup)\n");
    fprintf(stderr, "  -q <float>  learning rate decay: final fraction, at end of training (default = 1.0 (no decay))\n");
    fprintf(stderr, "  -c <float>  weight decay (default = 0.0f)\n");
    // evaluation
    fprintf(stderr, "  -v <int>    val_loss_every, how often we evaluate val loss (default = 20)\n");
    fprintf(stderr, "  -m <int>    val_max_steps, up to how many val batches to estimate val loss? (default = 20)\n");
    fprintf(stderr, "  -s <int>    sample_every, how often we inference the model (default = 20)\n");
    fprintf(stderr, "  -g <int>    genT, how many steps of inference we do (default = 64)\n");
    fprintf(stderr, "  -h <int>    hellaswag eval run? (default = 0)\n");
    // debugging
    fprintf(stderr, "  -a <int>    overfit a single batch? 0/1. useful for debugging\n");
    // numerics
    fprintf(stderr, "  -f <int>    enable_tf32 override (default: 1, set to 0 to disable tf32)\n");
    fprintf(stderr, "  -w <int>    keep f32 copy of weights for the optimizer? (default: 1)\n");
    // memory management
    fprintf(stderr, "  -z <int>    zero_stage, Zero Optimization Stage, 0,1,2,3 (default = 0)\n");
    fprintf(stderr, "  -r <int>    recompute: less memory but less speed. (default = 1), 0|1|2 = none,gelu,gelu+ln\n");
    exit(EXIT_FAILURE);
}

// ----------------------------------------------------------------------------
// main training loop
int main(int argc, char *argv[]) {
    multi_gpu_config = multi_gpu_config_init(&argc, &argv);

    // read in the (optional) command line arguments
    const char* train_data_pattern = "dev/data/tinyshakespeare/tiny_shakespeare_train.bin";
    const char* val_data_pattern = "dev/data/tinyshakespeare/tiny_shakespeare_val.bin";
    const char* load_filename = "gpt2_124M_bf16.bin"; // bf16 weights of the model
    const char* output_log_dir = NULL;
    int checkpoint_every = 0; // write optimization checkpoints every how many steps?
    int resume = 0; // resume the optimization, if one is found inside output_log_dir?
    int B = 4; // batch size
    int T = 1024; // sequence length max
    int total_batch_size = -1; // will be calculated down below later, if not provided
    float learning_rate = 3e-4f;
    int warmup_iterations = 0;
    float final_learning_rate_frac = 1.0f; // final fraction of learning rate, at end of training
    float weight_decay = 0.0f;
    int val_loss_every = 20; // every how many steps do we eval validation loss?
    int val_max_steps = 20; // how many batches max do we eval for validation loss?
    int sample_every = 20; // every how many steps to do inference?
    int genT = 64; // number of steps of inference we will do
    int overfit_single_batch = 0; // useful for debugging, 1 = only load a single data batch once
    int max_steps = -1;
    int override_enable_tf32 = 1;
    int use_master_weights = 1;
    int recompute = 1; // recompute during backward setting, 0 = none, 1 = recompute gelu
    int zero_stage = 0; // Zero Optimization Stage for Multi-GPU training
    int hellaswag_eval = 0;
    for (int i = 1; i < argc; i+=2) {
        if (i + 1 >= argc) { error_usage(); } // must have arg after flag
        if (argv[i][0] != '-') { error_usage(); } // must start with dash
        if (strlen(argv[i]) != 2) { error_usage(); } // must be -x (one dash, one letter)
        // read in the args
        if (argv[i][1] == 'i') { train_data_pattern = argv[i+1]; }
        else if (argv[i][1] == 'j') { val_data_pattern = argv[i+1]; }
        else if (argv[i][1] == 'e') { load_filename = argv[i+1]; }
        else if (argv[i][1] == 'o') { output_log_dir = argv[i+1]; }
        else if (argv[i][1] == 'n') { checkpoint_every = atoi(argv[i+1]); }
        else if (argv[i][1] == 'y') { resume = atoi(argv[i+1]); }
        else if (argv[i][1] == 'b') { B = atoi(argv[i+1]); } // Per-GPU (micro) batch size
        else if (argv[i][1] == 't') { T = atoi(argv[i+1]); }
        else if (argv[i][1] == 'd') { total_batch_size = atoi(argv[i+1]); }
        else if (argv[i][1] == 'l') { learning_rate = atof(argv[i+1]); }
        else if (argv[i][1] == 'u') { warmup_iterations = atoi(argv[i+1]); }
        else if (argv[i][1] == 'q') { final_learning_rate_frac = atof(argv[i+1]); }
        else if (argv[i][1] == 'c') { weight_decay = atof(argv[i+1]); }
        else if (argv[i][1] == 'x') { max_steps = atoi(argv[i+1]); }
        else if (argv[i][1] == 'v') { val_loss_every = atoi(argv[i+1]); }
        else if (argv[i][1] == 'm') { val_max_steps = atoi(argv[i+1]); }
        else if (argv[i][1] == 's') { sample_every = atoi(argv[i+1]); }
        else if (argv[i][1] == 'g') { genT = atoi(argv[i+1]); }
        else if (argv[i][1] == 'a') { overfit_single_batch = atoi(argv[i+1]); }
        else if (argv[i][1] == 'f') { override_enable_tf32 = atoi(argv[i+1]); }
        else if (argv[i][1] == 'w') { use_master_weights = atoi(argv[i+1]); }
        else if (argv[i][1] == 'z') { zero_stage = atoi(argv[i+1]); }
        else if (argv[i][1] == 'r') { recompute = atoi(argv[i+1]); }
        else if (argv[i][1] == 'h') { hellaswag_eval = atoi(argv[i+1]); }
        else { error_usage(); }
    }
    // should do a bit more error checking here
    assert(warmup_iterations >= 0);
    if (output_log_dir != NULL) {
        assert(strlen(output_log_dir) < 400); // careful bunch of hardcoded snprintf around this
    }
    // check if output_log_dir does not exist or is a file
    struct stat info;
    if (output_log_dir != NULL && (stat(output_log_dir, &info ) != 0 || !(info.st_mode & S_IFDIR))) {
        fprintf(stderr, "-o \"%s\" does not exist or is a file - are you specifying a file instead of dir?\n", output_log_dir);
        exit(EXIT_FAILURE);
    }
    int tokens_per_fwdbwd = B * T * multi_gpu_config.num_processes; // one micro-batch processes this many tokens
    // calculate sensible default for total batch size as assuming no gradient accumulation
    if (total_batch_size == -1) { total_batch_size = tokens_per_fwdbwd; }
    // calculate the number of gradient accumulation steps from the desired total batch size
    assert(total_batch_size % tokens_per_fwdbwd == 0);
    int grad_accum_steps = total_batch_size / tokens_per_fwdbwd;
    // if we're only overfitting a single batch for debugging, let's overfit the first batch
    // from val instead of train split, because val is smaller and faster. (train_gpt2.py does the same)
    if (overfit_single_batch == 1) { train_data_pattern = val_data_pattern; }
    printf0("+-----------------------+----------------------------------------------------+\n");
    printf0("| Parameter             | Value                                              |\n");
    printf0("+-----------------------+----------------------------------------------------+\n");
    printf0("| train data pattern    | %-50s |\n", train_data_pattern);
    printf0("| val data pattern      | %-50s |\n", val_data_pattern);
    printf0("| output log dir        | %-50s |\n", output_log_dir == NULL ? "NULL" : output_log_dir);
    printf0("| checkpoint_every      | %-50d |\n", checkpoint_every);
    printf0("| resume                | %-50d |\n", resume);
    printf0("| micro batch size B    | %-50d |\n", B);
    printf0("| sequence length T     | %-50d |\n", T);
    printf0("| total batch size      | %-50d |\n", total_batch_size);
    printf0("| learning rate (LR)    | %-50e |\n", learning_rate);
    printf0("| warmup iterations     | %-50d |\n", warmup_iterations);
    printf0("| final LR fraction     | %-50e |\n", final_learning_rate_frac);
    printf0("| weight decay          | %-50e |\n", weight_decay);
    printf0("| max_steps             | %-50d |\n", max_steps);
    printf0("| val_loss_every        | %-50d |\n", val_loss_every);
    printf0("| val_max_steps         | %-50d |\n", val_max_steps);
    printf0("| sample_every          | %-50d |\n", sample_every);
    printf0("| genT                  | %-50d |\n", genT);
    printf0("| overfit_single_batch  | %-50d |\n", overfit_single_batch);
    printf0("| use_master_weights    | %-50s |\n", use_master_weights ? "enabled" : "disabled");
    printf0("| recompute             | %-50d |\n", recompute);
    printf0("+-----------------------+----------------------------------------------------+\n");

    common_start(override_enable_tf32, false); // common init code for train/test/profile
    const char* precision_str = (PRECISION_MODE == PRECISION_FP32)
                              ? (cublas_compute == HIPBLAS_COMPUTE_32F_FAST_TF32 ? "TF32" : "FP32")
                              : (PRECISION_MODE == PRECISION_FP16 ? "FP16" : "BF16");
    printf0("| device                | %-50s |\n", deviceProp.name);
    printf0("| peak TFlops           | %-50.1f |\n", get_flops_promised(deviceProp.name, PRECISION_MODE));
    printf0("| precision             | %-50s |\n", precision_str);
    printf0("+-----------------------+----------------------------------------------------+\n");

    // figure out if we are going to be resuming the optimization
    char filename_buffer[512];
    int resuming = 0;
    int resume_max_step = find_max_step(output_log_dir);
    if (resume == 1) {
        // find the DONE file with the highest step count
        assert(output_log_dir != NULL);
        if (resume_max_step == -1) {
        } else {
            resuming = 1;
            snprintf(filename_buffer, 512, "%s/model_%08d.bin", output_log_dir, resume_max_step);
        }
    }

    // build the GPT-2 model
    GPT2 model;
    // if load_filename is of the form "dX" where X is an integer (e.g. d12), then we build
    // a random model with the depth of the model specified by X (e.g. 12). otherwise interpret
    // this variable as a checkpoint filename, and load that checkpoint
    assert(strlen(load_filename) >= 2);
    if (resuming == 1) {
        gpt2_build_from_checkpoint(&model, filename_buffer);
    } else if (load_filename[0] == 'd') {
        int depth = atoi(load_filename + 1);
        if (depth > 1 && depth <= 1000) { // we're not going to train models this big right? heh
            gpt2_build_from_random(&model, depth);
        } else {
            exit(EXIT_FAILURE);
        }
    } else {
        gpt2_build_from_checkpoint(&model, load_filename);
    }

    model.use_master_weights = use_master_weights;
    model.recompute = recompute;
    printf0("| weight init method    | %-50s |\n", resuming == 1 ? "intermediate checkpoint" : (load_filename[0] == 'd' ? "random" : "OpenAI's GPT-2 checkpoint"));
    printf0("| max_sequence_length T | %-50d |\n", model.config.max_seq_len);
    printf0("| vocab_size V          | %-50d |\n", model.config.vocab_size);
    printf0("| padded_vocab_size Vp  | %-50d |\n", model.config.padded_vocab_size);
    printf0("| num_layers L          | %-50d |\n", model.config.num_layers);
    printf0("| num_heads NH          | %-50d |\n", model.config.num_heads);
    printf0("| channels C            | %-50d |\n", model.config.channels);
    printf0("| num_parameters        | %-50zu |\n", model.num_parameters);
    printf0("+-----------------------+----------------------------------------------------+\n");

    // build DataLoaders for both train and val
    DataLoader train_loader, val_loader;
    dataloader_init(&train_loader, train_data_pattern, B, T, multi_gpu_config.process_rank, multi_gpu_config.num_processes);
    dataloader_init(&val_loader, val_data_pattern, B, T, multi_gpu_config.process_rank, multi_gpu_config.num_processes);
    // figure out the number of training steps we will run for
    int train_num_batches = max_steps; // passed in from command line
    if (train_num_batches == -1) {
        // sensible default is to train for exactly one epoch
        size_t ntok = train_loader.num_tokens;
        // the number of (outer loop) steps each process should take for us to reach one epoch
        train_num_batches = ntok / total_batch_size;
    }
    // figure out the number of validation steps to run for
    int val_num_batches = val_max_steps; // passed in from command line
    if (val_num_batches == -1) {
        // sensible default is to evaluate the full validation split
        size_t ntok = val_loader.num_tokens;
        // note that unlike the training loop, there is no gradient accumulation inner loop here
        val_num_batches = ntok / tokens_per_fwdbwd;
    }
    printf0("| train_num_batches     | %-50d |\n", train_num_batches);
    printf0("| val_num_batches       | %-50d |\n", val_num_batches);
    printf0("+-----------------------+----------------------------------------------------+\n");

    // build an EvalLoader for HellaSwag
    EvalLoader eval_loader;
    const char* hellaswag_path = "dev/data/hellaswag/hellaswag_val.bin";
    const char hellaswag_available = access(hellaswag_path, F_OK) == 0;
    const char run_hellaswag = hellaswag_eval && hellaswag_available;
    if (run_hellaswag) {
        evalloader_init(&eval_loader, hellaswag_path, B, T, multi_gpu_config.process_rank, multi_gpu_config.num_processes);
    }
    printf0("| run hellaswag         | %-50s |\n", run_hellaswag ? "yes" : "no");
    printf0("+-----------------------+----------------------------------------------------+\n");

    // pretty print in a table the multi-gpu configuration as well
    set_zero_configs(&multi_gpu_config, zero_stage, model.num_parameters);
    printf0("| num_processes         | %-50d |\n", multi_gpu_config.num_processes);
    printf0("| zero_stage            | %-50d |\n", multi_gpu_config.zero_stage);
    printf0("+-----------------------+----------------------------------------------------+\n");

    // prints outside of pretty table to here and below
    if (!hellaswag_available) {
        printf0("HellaSwag eval not found at %s, skipping its evaluation\n", hellaswag_path);
        printf0("You can run `python dev/data/hellaswag.py` to export and use it with `-h 1`.\n");
    }
    // more prints related to allocations from gpt2_build_from_checkpoint down here to not mess up our table above
    printf0("num_parameters: %zu => bytes: %zu\n", model.num_parameters, model.num_parameters_bytes);
    printf0("allocated %d MiB for model parameters\n", (int)round(model.num_parameters_bytes / (1024 * 1024)));
    // few more prints for gradient accumulation math up above
    printf0("batch_size B=%d * seq_len T=%d * num_processes=%d and total_batch_size=%d\n",
            B, T, multi_gpu_config.num_processes, total_batch_size);
    printf0("=> setting grad_accum_steps=%d\n", grad_accum_steps);

    // set up logging
    create_dir_if_not_exists(output_log_dir);
    Logger logger;
    logger_init(&logger, output_log_dir, multi_gpu_config.process_rank, resume);

    // set up the Tokenizer
    Tokenizer tokenizer;
    tokenizer_init(&tokenizer, "gpt2_tokenizer.bin");

    // some memory for generating samples from the model
    int* gen_tokens = (int*)mallocCheck(B * T * sizeof(int));
    floatX* cpu_logits_raw = (floatX*)mallocCheck(model.config.vocab_size * sizeof(floatX));
    float*  cpu_logits = (float*)mallocCheck(model.config.vocab_size * sizeof(float));

    // if we found a checkpoint to resume from, load the optimization state
    int step = 0;
    if (resuming == 1) {
        snprintf(filename_buffer, 512, "%s/state_%08d_%05d.bin", output_log_dir, resume_max_step, multi_gpu_config.process_rank);
        load_state(&step, &model, &train_loader, filename_buffer);
    }

    // train
    hipEvent_t start, end;
    cudaCheck(hipEventCreate(&start));
    cudaCheck(hipEventCreate(&end));
    cudaCheck(hipProfilerStart());
    double total_sum_iteration_time_s = 0.0;
    float ema_tokens_per_second = 0.0f;
    for (; step <= train_num_batches; step++) {
        NvtxRange step_range("Train step", step);

        int last_step = step == train_num_batches;

        // once in a while estimate the validation loss (all processes collaborate)
        if (step % val_loss_every == 0 || last_step) {
            NvtxRange validation_range("validation");
            float val_loss = 0.0f;
            dataloader_reset(&val_loader);
            for (int i = 0; i < val_num_batches; i++) {
                dataloader_next_batch(&val_loader);
                gpt2_forward(&model, val_loader.inputs, val_loader.targets, B, T);
                val_loss += model.mean_loss;
            }
            val_loss /= val_num_batches;
            val_loss = multi_gpu_cpu_float_sum(val_loss) / multi_gpu_config.num_processes;
            printf0("val loss %f\n", val_loss);
            logger_log_val(&logger, step, val_loss);
        }

        // once in a while estimate HellaSwag accuracy (all processes collaborate)
        if (run_hellaswag &&
           ((step > 0 && step % val_loss_every == 0) || last_step)) {
            NvtxRange evaluation_range("evaluation");
            float eval_acc_norm = 0.0f;
            evalloader_reset(&eval_loader);
            for (int i = 0; i < eval_loader.num_batches; i++) {
                if (i % 10 == 0) { printf("evaluating HellaSwag: %d/%d\r", i, eval_loader.num_batches); }
                evalloader_next_batch(&eval_loader);
                gpt2_forward(&model, eval_loader.inputs, eval_loader.targets, B, T);
                int correct = evalloader_stat_losses(&eval_loader, model.cpu_losses_fp32);
                eval_acc_norm += (float)correct;
            }
            // careful because not all ranks may have the exact same allocation of number of examples
            eval_acc_norm = multi_gpu_cpu_float_sum(eval_acc_norm);
            printf0("HellaSwag: %d/%d = %f\n", (int)eval_acc_norm, eval_loader.num_examples, eval_acc_norm / eval_loader.num_examples);
            logger_log_eval(&logger, step, eval_acc_norm / eval_loader.num_examples);
        }

        // once in a while do model inference to print generated text (only rank 0)
        if (multi_gpu_config.process_rank == 0 && sample_every > 0 &&
           (step > 0 && (step % sample_every) == 0 || last_step)) {
            NvtxRange generation_range("generation");
            unsigned long long sample_rng_state = 1337;
            // fill up gen_tokens with the <|endoftext|> token, which kicks off the generation
            int eot_token = tokenizer.eot_token;
            for(int i = 0; i < B * T; ++i) {
                gen_tokens[i] = eot_token;
            }
            // now sample from the model autoregressively
            printf("generating:\n---\n");
            for (int t = 1; t < genT; t++) {
                NvtxRange generation_range("Generation step", t);
                // note that inference is very wasteful here because for each token
                // we re-calculate the forward pass for all of (B,T) positions from scratch
                // but the inference here is just for sanity checking anyway
                // and we can maybe optimize a bit more later, with careful tests
                gpt2_forward(&model, gen_tokens, NULL, B, T);
                // furthermore, below we're only using b=0 (i.e. the first row) of all B rows
                // we're in principle running B "inference streams" in parallel here
                // only using position 0 because it's a bit faster (copy less probs from GPU -> CPU)
                // get the V-dimensional vector probs[0, t-1, :]
                floatX* logits = model.acts.output + (t - 1) * model.config.padded_vocab_size;
                // move probs back to CPU and sample (note we only move the first vocab_size logits, ignoring the padding)
                cudaCheck(hipMemcpy(cpu_logits_raw, logits, model.config.vocab_size * sizeof(floatX), hipMemcpyDeviceToHost));
                // convert to FP32 into cpu_logits (this does nothing useful if floatX == float)
                for (int i = 0; i < model.config.vocab_size; i++) {
                    cpu_logits[i] = (float)cpu_logits_raw[i];
                }
                // sample the next token
                float coin = random_f32(&sample_rng_state);
                int next_token = sample_softmax(cpu_logits, model.config.vocab_size, coin);
                gen_tokens[t] = next_token;
                // print the generated token, either using the Tokenizer or a fallback
                if (tokenizer.init_ok) {
                    const char* token_str = tokenizer_decode(&tokenizer, next_token);
                    safe_printf(token_str);
                } else {
                    // fall back to printing the token id
                    printf("%d ", next_token);
                }
                fflush(stdout);
            }
            printf("\n---\n");
        }

        // once in a while checkpoint the optimization state (all ranks)
        if ((checkpoint_every > 0 && output_log_dir != NULL && resuming == 0) &&
            ((step > 0 && step % checkpoint_every == 0) || last_step)) {
            assert(strlen(output_log_dir) < 400); // being a bit lazy here
            // only rank 0 writes the model file because it is the same across all ranks
            if (multi_gpu_config.process_rank == 0) {
                snprintf(filename_buffer, 512, "%s/model_%08d.bin", output_log_dir, step);
                gpt2_write_to_checkpoint(&model, filename_buffer);
            }
            // all ranks write their state file
            snprintf(filename_buffer, 512, "%s/state_%08d_%05d.bin", output_log_dir, step, multi_gpu_config.process_rank);
            save_state(filename_buffer, step, &model, &train_loader);
            // DONE file is a signal that this checkpoint as a whole is complete
            multi_gpu_barrier(&multi_gpu_config);
            if (multi_gpu_config.process_rank == 0) {
                snprintf(filename_buffer, 512, "%s/DONE_%08d", output_log_dir, step);
                FILE* done_file = fopenCheck(filename_buffer, "w");
                fclose(done_file);
            }
            multi_gpu_barrier(&multi_gpu_config);
        }
        resuming = 0;

        // bit confusing: we want to make sure to eval and sample on 0th iteration
        // but also after the very last iteration. so we loop for step <= train_num_batches
        // instead of just < train_num_batches (one extra due to <=), only to do
        // the validation/sampling one last time, and then we break right here as we're done.
        if (last_step) { break; }

        // --------------- TRAINING SECTION BEGIN -----------------
        // do one training step, doing forward/backward/update on total_batch_size tokens
        hipEventRecord(start);
        // gradient accumulation loop over micro-batches
        float lossf = 0.0f; // for getting the mean loss over the accumulation steps
        for (int micro_step = 0; micro_step < grad_accum_steps; micro_step++) {
            // fetch the next data batch
            // and if we're overfitting a single batch, we'll only call this a single time
            if (overfit_single_batch == 0 ||
               (overfit_single_batch == 1 && step == 0 && micro_step == 0)) {
                dataloader_next_batch(&train_loader);
            }
            // forward pass. note that we pass in grad_accum_steps, which scales down the loss
            gpt2_forward(&model, train_loader.inputs, train_loader.targets, B, T, grad_accum_steps);
            lossf += model.mean_loss; // the mean_loss was normalized by grad_accum_steps inside gpt2_forward
            // backward pass. all model params accumulate gradients with += inside this inner loop
            gpt2_backward(&model, train_loader.inputs);
        }
        // override the mean loss, accounting for the gradient accumulation loop
        // this is esp important to do here in multigpu update below, where model.mean_loss gets allreduced
        model.mean_loss = lossf;
        // average the loss and the gradients between all processes
        gpt2_multi_gpu_loss_and_grad_reduce(&model, &multi_gpu_config);
        // learning rate schedule: warmup linearly to max LR, then cosine decay to LR * final_learning_rate_frac
        float step_learning_rate = learning_rate;
        if (step < warmup_iterations) {
            step_learning_rate = learning_rate * ((float)(step + 1)) / warmup_iterations;
        } else {
            float decay_ratio = ((float)(step - warmup_iterations)) / (train_num_batches - warmup_iterations);
            assert(0.0f <= decay_ratio && decay_ratio <= 1.0f);
            float coeff = 0.5f * (1.0f + cosf(M_PI * decay_ratio)); // coeff starts at 1 and goes to 0
            assert(0.0f <= coeff && coeff <= 1.0f);
            float min_lr = learning_rate * final_learning_rate_frac;
            step_learning_rate = min_lr + coeff * (learning_rate - min_lr);
        }
        // update the model parameters
        float grad_norm = gpt2_update(&model, step_learning_rate, 0.9f, 0.95f, 1e-8f, weight_decay, 1.0f, step+1, &multi_gpu_config);
        gpt2_multi_gpu_param_gather(&model, &multi_gpu_config);
        // zero out the gradients for the next iteration
        gpt2_zero_grad(&model);
        cudaCheck(hipEventRecord(end));
        cudaCheck(hipEventSynchronize(end)); // wait for the end event to finish to get correct timings
        // --------------- TRAINING SECTION END -------------------
        // everything that follows now is just diagnostics, prints, logging, etc.

        // todo - move or double-buffer all of this timing logic to avoid idling the GPU at this point!
        float time_elapsed_ms;
        cudaCheck(hipEventElapsedTime(&time_elapsed_ms, start, end));
        size_t tokens_processed = (size_t)multi_gpu_config.num_processes * B * T * grad_accum_steps;
        float tokens_per_second = tokens_processed / time_elapsed_ms * 1000.0f;
        float bias_corrected_ema_tokens_per_second = tokens_per_second; // by default set to non-ema version
        if (step > 0) { // consider the first batch to be a warmup (e.g. cuBLAS/cuDNN initialisation)
            total_sum_iteration_time_s += time_elapsed_ms / 1000.0f;
            // smooth out the tok/s with an exponential moving average, and bias correct just like in AdamW
            ema_tokens_per_second = 0.95f * ema_tokens_per_second + 0.05f * tokens_per_second;
            bias_corrected_ema_tokens_per_second = ema_tokens_per_second / (1.0f - powf(0.95f, step));
        }
        float accumulated_loss = multi_gpu_config.num_processes == 1 ? model.mean_loss : model.accumulated_mean_loss;
        float mfu = gpt2_estimate_mfu(&model, B * T * grad_accum_steps, time_elapsed_ms / 1000.0f);
        printf0("step %4d/%d | train loss %7.6f | norm %6.4f | lr %.2e | %.2f ms | %.1f%% bf16 MFU | %.0f tok/s\n",
                step + 1, train_num_batches, accumulated_loss, grad_norm, step_learning_rate,
                time_elapsed_ms, 100*mfu, bias_corrected_ema_tokens_per_second);
        logger_log_train(&logger, step, model.mean_loss, step_learning_rate, grad_norm);

        // disable the profiler after 3 steps of optimization
        if (step == 3) { hipProfilerStop(); }
    }
    // add a total average, for optimizations that are only mild improvements (excluding 1st batch as warmup)
    printf0("total average iteration time: %f ms\n", total_sum_iteration_time_s / (train_num_batches-1) * 1000);

    // free and destroy everything
    cudaCheck(hipEventDestroy(end));
    cudaCheck(hipEventDestroy(start));
    if (run_hellaswag) { evalloader_free(&eval_loader); }
    dataloader_free(&train_loader);
    dataloader_free(&val_loader);
    tokenizer_free(&tokenizer);
    free(cpu_logits_raw);
    free(cpu_logits);
    free(gen_tokens);
    multi_gpu_config_free(&multi_gpu_config);
    gpt2_free(&model);
    common_free(model);
    return 0;
}
#endif
